#include "hip/hip_runtime.h"

#include "string.h"
#include "stdio.h"
#include "stdlib.h"
#include <chrono>
#include <iostream>
#include <iomanip>
#include <climits>

using namespace std;

#define PRINTLINEMACRO {	\
	printf("%d in %s\n", __LINE__, __FILE__); \
}

__global__ void gpu_mat_mul(float *A, float* B, float* C, long N)
{
  long i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= N) return;
  float val = 0.0;
  float* a = A + (i / N);
  float* b = B + (i % N);
  for(long j = 0; j < N; j++){
  	val += *(a + j) * *(b + (N*j));
  }
  *(C + i) = val;
}

int main(int argc, char **argv){
	auto begin = chrono::high_resolution_clock::now();
	if(argc < 2){
		printf("usage: ./%s <Size> \n", argv[0]);
		return 1;
	}
    
   	long size = atol(argv[1]);
   	long N = size*size;
   	size_t bytes = sizeof(float) * N;
   	
   	float* A_h = (float*)malloc(bytes);
   	float* B_h = (float*)malloc(bytes);
   	float* C_h = (float*)malloc(bytes);
   	
   	memset(A_h, 1.0, bytes);
   	memset(B_h, 1.0, bytes);
   	
	float *A_d, *B_d, *C_d;
	hipMalloc((void**)&A_d, bytes);
	hipMalloc((void**)&B_d, bytes);
	hipMalloc((void**)&C_d, bytes);
	
	hipMemcpy(A_d, A_h, bytes, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, bytes, hipMemcpyHostToDevice);
	free(A_h);
	free(B_h);
	
	long threadsPerBlock = 32;
	long numBlocks = (N % threadsPerBlock == 0) ? N/threadsPerBlock : (N/threadsPerBlock)+1;
	
	gpu_mat_mul <<< numBlocks, threadsPerBlock >>> (A_d, B_d, C_d, N);
	
	hipDeviceSynchronize();

	hipMemcpy(C_h, C_d, bytes, hipMemcpyDeviceToHost);
	
	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);
	
	free(C_h);
	
	auto end = chrono::high_resolution_clock::now();
	auto duration = chrono::duration_cast<chrono::milliseconds>(end - begin);
	cout << "Number of threads: " << setw(2) << threadsPerBlock * numBlocks << " Matrix size: " << setw(9) << size;
	cout << " Milliseconds taken: " << setw(15) << duration.count() << endl;
	return 0;
}