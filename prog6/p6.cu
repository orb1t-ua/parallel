#include "hip/hip_runtime.h"

#include "string.h"
#include "stdio.h"
#include "stdlib.h"
#include <chrono>
#include <iostream>
#include <iomanip>

using namespace std;


__global__ void gpu_mat_mul(float *A, float* B, float* C, long N)
{
  long i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= N) return;
  float val = 0.0;
  float* a = A + (i / N);
  float* b = B + (i % N);
  for(long j = 0; j < N; j++){
  	val += *(a + j) * *(b + (N*j));
  }
  *(C + i) = val;
}

int main(int argc, char **argv){
	auto begin = chrono::high_resolution_clock::now();
	if(argc < 2){
		printf("usage: ./%s <Size> \n", argv[0]);
		return 1;
	}
    hipSetDevice(0);
   	long size = atol(argv[1]);
   	long N = size*size;
   	size_t bytes = sizeof(float) * N;
   	
   	float* A_h = (float*)malloc(bytes);
   	float* B_h = (float*)malloc(bytes);
   	float* C_h = (float*)malloc(bytes);
   	
	float *A_d, *B_d, *C_d;
	hipMalloc((void**)&A_d, bytes);
	hipMalloc((void**)&B_d, bytes);
	hipMalloc((void**)&C_d, bytes);
	
	memset(A_h, 1.0, bytes);
	memset(B_h, 1.0, bytes);
	
	hipMemcpy(A_d, A_h, bytes, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, bytes, hipMemcpyHostToDevice);
	free(B_h);
	free(A_h);
	
	int blockSize;   // The launch configurator returned block size 
	int minGridSize; // The minimum grid size needed to achieve the 
		             // maximum occupancy for a full device launch 
	int gridSize;    // The actual grid size needed, based on input size 
	hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, gpu_mat_mul, 0, N);
	// Round up according to array size 
	gridSize = (N + blockSize - 1) / blockSize;
	gpu_mat_mul <<< gridSize, blockSize >>> (A_d, B_d, C_d, N);
	
	hipMemcpy(C_h, C_d, bytes, hipMemcpyDeviceToHost);
	
	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);
	
	free(C_h);

	auto end = chrono::high_resolution_clock::now();
	auto duration = chrono::duration_cast<chrono::milliseconds>(end - begin);
	cout << "Number of threads: " << setw(2) << blockSize * gridSize << " Matrix size: " << setw(9) << N;
	cout << " Microseconds taken: " << setw(15) << duration.count() << endl;
	return 0;
}
