#include "hip/hip_runtime.h"

#include "string.h"
#include "stdio.h"
#include "stdlib.h"
#include <chrono>
#include <iostream>
#include <iomanip>
#include <climits>

using namespace std;

#define PRINTLINEMACRO {	\
	printf("%d in %s\n", __LINE__, __FILE__); \
}

__global__ void gpu_mat_mul(float* A, float* B, float* C, long width, long N)
{
  const long i = (blockIdx.x * blockDim.x + threadIdx.x) % N;
  float val = 0.0;
  float* a = A + (i / width);
  float* b = B + (i % width);
  for(long j = 0; j < width; j++){
  	val += *(a + j) * *(b + (width*j));
  }
  *(C + i) = val;
}

int main(int argc, char **argv){
	auto begin = chrono::high_resolution_clock::now();
	if(argc < 2){
		printf("usage: ./%s <Size> \n", argv[0]);
		return 1;
	}
    
   	long size = atol(argv[1]);
   	long N = size*size;
   	size_t bytes = sizeof(float) * N;
   	
   	float* A_h = (float*)malloc(bytes);
   	float* B_h = (float*)malloc(bytes);
   	float* C_h = (float*)malloc(bytes);
   	
   	memset(A_h, 1.0, bytes);
   	memset(B_h, 1.0, bytes);
   	
	float *A_d, *B_d, *C_d;
	hipMalloc((void**)&A_d, bytes);
	hipMalloc((void**)&B_d, bytes);
	hipMalloc((void**)&C_d, bytes);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Error: %s\n", hipGetErrorString(err));
	
	hipMemcpy(A_d, A_h, bytes, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, bytes, hipMemcpyHostToDevice);
	
	err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Error: %s\n", hipGetErrorString(err));
	
	free(A_h);
	free(B_h);
	
	long threadsPerBlock = 16;
	long numBlocks = (N % threadsPerBlock == 0) ? N/threadsPerBlock : (N/threadsPerBlock)+1;
	
	gpu_mat_mul <<< numBlocks, threadsPerBlock >>> (A_d, B_d, C_d, size, N);
	hipDeviceSynchronize();
	
	err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Error: %s\n", hipGetErrorString(err));
	
	hipMemcpy(C_h, C_d, bytes, hipMemcpyDeviceToHost);
	
	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);
	
	err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Error: %s\n", hipGetErrorString(err));
	
	free(C_h);
	
	auto end = chrono::high_resolution_clock::now();
	auto duration = chrono::duration_cast<chrono::milliseconds>(end - begin);
	cout << "Number of threads: " << setw(2) << threadsPerBlock * numBlocks << " Matrix size: " << setw(9) << size;
	cout << " Milliseconds taken: " << setw(15) << duration.count() << endl;
	return 0;
}
