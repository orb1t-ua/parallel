#include "hip/hip_runtime.h"

#include "string.h"
#include "stdio.h"
#include "stdlib.h"
#include <chrono>
#include <iostream>
#include <iomanip>

using namespace std;

#define CUDA_DEBUG {	\
		hipError_t err = hipGetLastError();	\
		if (err != hipSuccess){ 	\
			printf("Error at %s %s: %s\n", __LINE__, __FILE__, hipGetErrorString(err));	\
			exit(1);	\
		}	\
}	

__global__ void gpu_mat_mul(float* A, float* B, float* C, long width, long N)
{
  const long i = (blockIdx.x * blockDim.x + threadIdx.x);
  if(i >= N) return;
  float val = 0.0;
  float* a = A + (i / width);
  float* b = B + (i % width);
  for(long j = 0; j < width; j++){
  	val += *(a + j) * *(b + (width*j));
  }
  *(C + i) = val;
}

int main(int argc, char **argv){
	auto begin = chrono::high_resolution_clock::now();
	if(argc < 2){
		printf("usage: ./%s <size> \n", argv[0]);
		return 1;
	}
    
   	long size = atol(argv[1]);
   	long N = size*size;
   	size_t bytes = sizeof(float) * N;
   	
   	float* A_h = (float*)malloc(bytes);
   	float* B_h = (float*)malloc(bytes);
   	float* C_h = (float*)malloc(bytes);
   	
   	for(long i = 0; i < N; i++){
   		*(A_h + i) = (float)i / (float)N;
   		*(B_h + i) = (float)i / (float)N;
   	}
   	
	float *A_d, *B_d, *C_d;
	hipMalloc((void**)&A_d, bytes);
	hipMalloc((void**)&B_d, bytes);
	hipMalloc((void**)&C_d, bytes);
	
	CUDA_DEBUG
	
	hipMemcpy(A_d, A_h, bytes, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, bytes, hipMemcpyHostToDevice);
	
	hipDeviceSynchronize();
	
	CUDA_DEBUG
	
	free(A_h);
	free(B_h);
	
	long threadsPerBlock = 1024;
	long numBlocks = N / threadsPerBlock;
	numBlocks += (N % threadsPerBlock > 0) ?  1 : 0;
	
	gpu_mat_mul <<< numBlocks, threadsPerBlock >>> (A_d, B_d, C_d, size, N);
	hipDeviceSynchronize();
	
	CUDA_DEBUG
	
	hipMemcpy(C_h, C_d, bytes, hipMemcpyDeviceToHost);
	
	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);
	
	CUDA_DEBUG
	
	free(C_h);
	
	auto end = chrono::high_resolution_clock::now();
	auto duration = chrono::duration_cast<chrono::milliseconds>(end - begin);
	cout << "Blocksize * Gridsize: " << setw(2) << threadsPerBlock * numBlocks << " Matrix size: " << setw(9) << size;
	cout << " Milliseconds taken: " << setw(15) << duration.count() << endl;
	return 0;
}
